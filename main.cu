#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "Mesher.h"
#include "Solver_sac.h"
#include "Obj_cylinder.h"
#include "Obj_rectangle.h"
#include "Obj_template.h"
#include <TNL/Timer.h>
#include <TNL/Logger.h>

using DeviceType = TNL::Devices::Host;
using DeviceTypeHost = TNL::Devices::Host;

using RealType = double;



int main()
{
    const double L = 0.1;               //[m]
    const int Nx = 300;                 //[1]
    const int Ny = 100;                 //[1]

    const double rho=1000;              //[kg/m3]
    const double ny=10e-5;              //[m2/s]

    const double ux=0.100;              //[m/s]
    const double ux_guess=0.1;          //[m/s]
    const double uy=0.000;              //[m/s]
    const double u_max_lattice =0.1;  //[0]

    const double Fx = 10;               //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)
    const double Fy = 0.0;              //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)

    const double time =1;               //[s]  
    const double plot_every=0.01;       //[s]

    int plot_every_it;
    int iterations;

    
    
    Mesher<RealType, DeviceTypeHost > mesh_rectangle(Ny,Nx);   

    //objects !! pristup od 0 !! horní index o 1 mensi je to 

    Obj_rectangle lower_wall( -1.0, Nx , -1.0, -1.0);
    Obj_rectangle upper_wall( -1.0, Nx, Ny , Ny );
    Obj_rectangle inlet(-1, -1, 0, Ny-1 );
    Obj_rectangle outlet(Nx , Nx, 0, Ny-1);
    Obj_cylinder cylinder(Ny/5, Nx/4,Ny/2+0.05*Ny);

    // MESH - structured bolean values of BC
    // 0 = solid | 1 = fluid | 2 = primitive inlet vertical | 3 = outlet (rho=1, right) | 4 = moving wall up | 5 = moving wall down | 6 = outlet (rh=1, left)

    mesh_rectangle.meshing(lower_wall,0);
    mesh_rectangle.meshing(upper_wall,0);
    mesh_rectangle.meshing(cylinder, 0);
    mesh_rectangle.meshing(outlet, 3);
    mesh_rectangle.meshing_moving(inlet, ux, 0, 2);

    //output mesh
    mesh_rectangle.output_VTK();

    Solver_sac<RealType, DeviceType> solver(Ny,Nx,mesh_rectangle);
    solver.convert_to_lattice(L, ux_guess, rho, ny, u_max_lattice);

    
    plot_every_it = std::ceil(plot_every/solver.Ct_pub);
    std::cout<<"\nPlotting every " << plot_every_it << " iterations.\n";
    iterations = std::ceil(time/solver.Ct_pub);
    std::cout<<"\nCalculation will run for "<<iterations<<" iterations.\n";
    
    solver.initialization_eq(rho, ux, uy, Fx, Fy, 0);

    solver.output_VTK_lattice();
    solver.output_VTK(0,plot_every_it);
    
    //solver run
    
    Timer timer;
    Logger logger(50, std::cout);

    timer.start();

    int k = 0;
    while(k<iterations) //err>=10e-4)
    {
        k++;
        solver.collision();
        solver.streaming();
        solver.bounce_back();
        solver.postpro();

        if(k%500==0 && k!=0)
        {
            //solver.Err();
            //printf("\n err=%e ux_center=%e uy_center=%e rho_center=%e k=%d\n",solver.err,solver.ux(Ny/2,Nx/2),solver.uy(Ny/2,Nx/2),solver.rho(Ny/2,Nx/2), k);
            solver.output_VTK_lattice();
             if (std::isnan(solver.err)) 
             {
                std::cout << "\n Error is NaN, breaking out.\n";
                break;
             }
        }

        

        if(k%plot_every_it==0)
        {
            solver.output_VTK(k,plot_every_it);
        }
   
    }

    timer.stop();
    
    timer.writeLog( logger, 0 );

    return 0;
}

