#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "./src/geo/Mesher.h"
#include "./src/sol/Solver_sac.h"
#include "./src/sol/Solver_LES.h"
#include "./src/geo/Obj_cylinder.h"
#include "./src/geo/Obj_rectangle.h"
#include "./src/geo/Obj_template.h"
#include <TNL/Timer.h>
#include <TNL/Logger.h>

using DeviceType = TNL::Devices::Cuda; //sac
using DeviceTypeHost = TNL::Devices::Host;

using RealType = float;



int main()
{
    const RealType L = 0.1f;                //[m]
    const int Nx = 400;                     //[1]
    const int Ny = 100;                     //[1]

    const RealType rho=1000.f;              //[kg/m3]
    const RealType ny=10e-5f;               //[m2/s]

    const RealType ux=10.f;               //[m/s] // 0.01 ok 0.1 fail
    const RealType ux_guess=15.f;          //[m/s]
    const RealType uy=0.f;                  //[m/s]
    const RealType u_max_lattice =0.1f;    //[0]

    const RealType Fx = 0.f;               //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)
    const RealType Fy = 0.0f;               //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)

    const RealType time =1.f;               //[s]
    const RealType plot_every=0.1f;         //[s]

    int plot_every_it;
    int iterations;

    
    
    Mesher<RealType, DeviceTypeHost > mesh_rectangle(Ny,Nx);   

    //objects !! pristup od 0 !! horní index o 1 mensi je to 

    Obj_rectangle lower_wall( -1.f, Nx , -1.f, -1.f);
    Obj_rectangle upper_wall( -1.f, Nx, Ny , Ny );
    Obj_rectangle inlet(-1, -1, 0, Ny-1 );
    Obj_rectangle outlet(Nx , Nx, 0, Ny-1);
    //Obj_cylinder cylinder(Ny/5, Nx/4,Ny/2+0.05f*Ny);
    Obj_rectangle blockage(-1, Nx/5, -1, 3*Ny/5);

    // MESH - structured bolean values of BC
    // 0 = solid | 1 = fluid | 2 = primitive inlet vertical | 3 = outlet (rho=1, right) | 4 = moving wall up | 5 = moving wall down | 6 = outlet (rh=1, left)

    mesh_rectangle.meshing(lower_wall,0);
    mesh_rectangle.meshing(upper_wall,0);
    //mesh_rectangle.meshing(cylinder, 0);
    mesh_rectangle.meshing(outlet, 3);
    mesh_rectangle.meshing_moving(inlet, ux, 0, 2);
    mesh_rectangle.meshing(blockage,0);

    //output mesh
    mesh_rectangle.output_VTK();

    Solver_LES<RealType, DeviceType> solver(Ny,Nx,mesh_rectangle);
    solver.convert_to_lattice(L, ux_guess, rho, ny, u_max_lattice);

    
    plot_every_it = std::ceil(plot_every/solver.Ct_pub);
    std::cout<<"\nPlotting every " << plot_every_it << " iterations.\n";
    iterations = std::ceil(time/solver.Ct_pub);
    std::cout<<"\nCalculation will run for "<<iterations<<" iterations.\n";

    plot_every_it = 500;
    
    solver.initialization_eq(rho, ux, uy, Fx, Fy, 0);

    solver.output_VTK_lattice();
    solver.output_VTK(0,plot_every_it);


    //solver run
    
    Timer timer_loop;
    Timer timer_collision;
    Timer timer_streaming;
    Timer timer_bounceback;
    Timer timer_postpro;
    Timer timer_err;
    Timer timer_output;

    Logger logger(50, std::cout);

    timer_loop.start();


    int k = 0;
    while(k<iterations) //err>=10e-4)
    {
        k++;
        timer_postpro.start();
        solver.omegaLES();
        timer_postpro.stop();
        
        timer_collision.start();
        solver.collision();
        timer_collision.stop();


        timer_streaming.start();
        solver.streaming();
        timer_streaming.stop();

        timer_bounceback.start();
        solver.bounce_back();
        timer_bounceback.stop();

        timer_postpro.start();
        solver.postpro();
        timer_postpro.stop();

        if(k%500==0 && k!=0)
        {

            timer_err.start();
            solver.Err();
            //printf("\n err=%e ux_center=%e uy_center=%e rho_center=%e k=%d\n",solver.err,solver.ux.getView()(Ny/2,Nx/2),solver.uy.getView()(Ny/2,Nx/2),solver.rho.getView()(Ny/2,Nx/2), k);
            printf("\n err=%e, k=%d \n" ,solver.err, k);
            if (std::isnan(solver.err))
             {
                std::cout << "\n Error is NaN, breaking out.\n";
                break;
             }

            timer_err.stop();
        }

        

        if(k%plot_every_it==0)
        {   
            timer_output.start();
            solver.output_VTK(k,plot_every_it);
            timer_output.stop();
        }
   
    }

    timer_loop.stop();
    
    logger.writeHeader("Timing of sections");
    //logger.writeSystemInformation(true);
    timer_loop.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_collision.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_streaming.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_bounceback.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_postpro.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_err.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_output.writeLog( logger, 0 );


    return 0;
}

