#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "./src/geo/Mesher.h"
#include "./src/sol/Solver_sac.h"
#include "./src/sol/Solver_LES.h"
#include "./src/geo/Obj_cylinder.h"
#include "./src/geo/Obj_rectangle.h"
#include "./src/geo/Obj_template.h"
#include <TNL/Timer.h>
#include <TNL/Logger.h>

using DeviceType = TNL::Devices::Cuda; //sac
using DeviceTypeHost = TNL::Devices::Host;

using RealType = float;



int main()
{
    const RealType L = 15.0f;                //[m]
    const int Nx = 3000;                     //[1]
    const int Ny = 400;                     //[1]

    const RealType rho=1000.f;              //[kg/m3]
    const RealType ny=1e-6f;               //[m2/s]

    const RealType ux=1e-4f;               //[m/s] // 0.01 ok 0.1 fail
    const RealType ux_guess=2e-4f;          //[m/s]
    const RealType uy=0.f;                  //[m/s]
    const RealType u_max_lattice =0.09f;    //[0]

    const RealType Fx = 0.0f;               //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)
    const RealType Fy = 0.0f;               //[kg/m2/s2]  <- force density (3rd dimension in 2D is equal to 1)

    const RealType time =10000000.f;               //[s]
    const RealType plot_every=200.f;         //[s]

    int plot_every_it;
    int iterations;

    bool run_simulation = true;
    
    
    Mesher<RealType, DeviceTypeHost > mesh_rectangle(Ny,Nx);   

    //objects !! pristup od 0 !! horní index o 1 mensi je to 

    Obj_rectangle lower_wall( -1.f, Nx , -1.f, -1.f);
    Obj_rectangle upper_wall( -1.f, Nx, Ny , Ny );
    Obj_rectangle front_wall(-1, -1, 0, Ny-1 );
    Obj_rectangle back_wall(Nx , Nx, 0, Ny-1);
    Obj_rectangle inlet(0, 0, 0, Ny-1 );
    Obj_rectangle outlet(Nx-1 , Nx-1, 0, Ny-1);
    //Obj_cylinder cylinder(Ny/5, Nx/4,Ny/2+0.05f*Ny);
    Obj_rectangle blockage(-1, Nx/5, -1, 3*Ny/5);

    // MESH - structured bolean values of BC
    // 0 = solid | 1 = fluid | 2 = primitive inlet vertical | 3 = outlet (rho=1, right) | 4 = moving wall up | 5 = moving wall down | 6 = outlet (rh=1, left)

    mesh_rectangle.meshing(lower_wall,0);
    mesh_rectangle.meshing(upper_wall,0);
    mesh_rectangle.meshing(front_wall,0);
    mesh_rectangle.meshing(back_wall,0);
    mesh_rectangle.meshing(outlet, 3);
    mesh_rectangle.meshing_moving(inlet, ux, 0, 2);

    //mesh_rectangle.meshing(cylinder, 0);
    mesh_rectangle.meshing(blockage,0);

    //output mesh
    mesh_rectangle.output_VTK();

    Solver_sac<RealType, DeviceType> solver(Ny,Nx,mesh_rectangle);
    solver.convert_to_lattice(L, ux_guess, rho, ny, u_max_lattice);

    
    plot_every_it = std::ceil(plot_every/solver.Ct_pub);
    std::cout<<"\nPlotting every " << plot_every_it << " iterations.\n";
    iterations = std::ceil(time/solver.Ct_pub);
    std::cout<<"\nCalculation will run for "<<iterations<<" iterations.\n";

    plot_every_it = 500;
    
    solver.initialization_eq(rho, ux, uy, Fx, Fy, 0);

    solver.output_VTK_lattice();
    solver.output_VTK(0,plot_every_it);


    //solver run
    
    Timer timer_loop;
    Timer timer_collision;
    Timer timer_streaming;
    Timer timer_bounceback;
    Timer timer_postpro;
    Timer timer_err;
    Timer timer_output;

    Logger logger(50, std::cout);

    timer_loop.start();


    int k = 0;
    while(k<iterations && run_simulation) //err>=10e-4)
    {
        k++;
        /*timer_postpro.start();
        solver.omegaLES();
        timer_postpro.stop();*/
        
        timer_collision.start();
        solver.collision();
        timer_collision.stop();


        timer_streaming.start();
        solver.streaming();
        timer_streaming.stop();

        timer_bounceback.start();
        solver.bounce_back();
        timer_bounceback.stop();

        timer_postpro.start();
        solver.postpro();
        timer_postpro.stop();

        if(k%500==0 && k!=0)
        {

            timer_err.start();
            solver.Err();
            //printf("\n err=%e ux_center=%e uy_center=%e rho_center=%e k=%d\n",solver.err,solver.ux.getView()(Ny/2,Nx/2),solver.uy.getView()(Ny/2,Nx/2),solver.rho.getView()(Ny/2,Nx/2), k);
            printf("\n err=%e, k=%d \n" ,solver.err, k);
            if (std::isnan(solver.err))
             {
                std::cout << "\n Error is NaN, breaking out.\n";
                break;
             }

            timer_err.stop();
        }

        

        if(k%plot_every_it==0)
        {   
            timer_output.start();
            solver.output_VTK(k,plot_every_it);
            timer_output.stop();
        }
   
    }

    timer_loop.stop();
    
    logger.writeHeader("Timing of sections");
    //logger.writeSystemInformation(true);
    timer_loop.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_collision.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_streaming.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_bounceback.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_postpro.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_err.writeLog( logger, 0 );
    logger.writeSeparator();
    timer_output.writeLog( logger, 0 );


    return 0;
}

