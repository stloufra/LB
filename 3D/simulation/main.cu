#include "hip/hip_runtime.h"
#pragma hd_warning_disable
#pragma nv_exec_check_disable


#include <iostream>
#include <TNL/Timer.h>
#include <TNL/Logger.h>

#include "src/geometry/geometryMesherBoundary.h"
#include "src/geometry/geometryObjectCuboid.h"
#include "src/solvers/SolverTurbulentLES.h"
#include "src/traits/LBMTraits.h"
#include "src/postprocesors/outputerVTK.h"
#include "src/postprocesors/outputerMesh.h"

#include "src/solvers/models/D3Q27/D3Q27.h"
#include "src/solvers/models/D3Q19/D3Q19.h"
#include "src/solvers/models/D3Q15/D3Q15.h"


using namespace TNL;
using DeviceType = typename LBMTraits::DeviceType;
using VectorType = typename LBMTraits::VectorType;
using RealType = typename LBMTraits::RealType;
using DeviceTypeHost = typename LBMTraits::DeviceTypeHost;
using LBMDataPointer = TNL::Pointers::SharedPointer<LBMData, DeviceType>;
using LBMConstantsPointer = TNL::Pointers::SharedPointer<LBMConstants, DeviceType>;

int main() {
    // https://volkov.eng.ua.edu/

    //------------------------INITIALIZATION--------------------------//


    // initialize data carrier objects
    LBMConstantsPointer Constants;
    LBMDataPointer Data;

    // model types selection
    using Model = D3Q27;

    using Initialisation        = InitializationEquilibriumConstVector<Model>;
    using Collision             = CollisionCumD3Q27TurbulentCombined<Model>;
    using Streaming             = StreamingAB<Model>;
    using BounceBackWall        = BounceBackWallHalf<Model>;
    using Inlet                 = InletVelocityMovingWall<Model>;
    using Outlet                = OutletNeighbourEquilibriumOmega<Model>;
    using Moments               = MomentDensityVelocityN27<Model>;  // SAME AS MODEL NUMBER
    using Error                 = ErrorQuadratic<Model>;
    using Turbulence            = OmegaLES<Model>;
    using NonDim                = NonDimensiolnaliseFactorsVelocity<Model>;
    using TimeAvg               = MomentTimeAvg<Model>;


    //initialize timers
    Timer timerMeshingBoundary;
    Logger logger(50, std::cout);

    //initialize methodical classes
    geometryMesherBoundary Mesher(Constants,
                                  Data);

    SolverTurbulentLES< Model,
            Initialisation,
            Collision,
            Streaming,
            BounceBackWall,
            Inlet,
            Outlet,
            Moments,
            Turbulence,
            Error,
            NonDim,
            TimeAvg> Solver( Constants,
                            Data);

    bool runSim = true;


    //------------------------DATA IN--------------------------//

    //set simulation initialization
    VectorType Init(0.f, 0.f, 3.f); //change to 1 in z
    Constants->VelocityInit = Init;


    //set meshing data
    Constants->resolution_factor = 0.1;
    Constants->additional_factor = 6;                              // at least 1 for additional wall around
    Constants->point_outside = {2, 1000, 0};
    Constants->file_name = "Fany.off";


    //set geometry objects

    //resolution 3
    geometryObjectCuboid cuboidInlet({150.f, 350.f, -8.f},
                                      {150.f, 350.f, 408.f},
                                      {155.f, 770.f, 408.f},-1);


    geometryObjectCuboid cuboidOutlet({3148.f, 345.f, -8.f},
                                      {3148.f, 345.f, 408.f},
                                      {3156.f, 770.f, 408.f},
                                      -2);

    VectorType VelocityInlet(5.f, 0.f, 0.f);


    VectorType NormalInlet(-1.f, 0.f, 0.f);

    VectorType NormalOutlet(1.f, 0.f, 0.f);

    //inlet parabolic data

    d3 inletCenter = {153.629 , 556.456, 300};
    RealType inletDimX = 0.f;
    RealType inletDimY = 400.f;
    RealType inletDimZ = 200.f;
    RealType meanVelocityInlet = 5.5f;       // 5

    //dumping tau outlet data
    Constants -> omegaDumpingLow = 0.f;
    Constants -> omegaDumpingHigh = 0.05f; // tau(Re=1000) = 0.55 -> 0.2


    //set physical data
    Constants->rho_fyz = 1.293f;                      //[kg/m3]     1000
    Constants->ny_fyz = 10e-5f;                       //[m2/s]
    Constants->u_guess_fyz = 5.5f;                    //[m/s] //TODO should be automatically calculated //5.5f
    Constants->Fx_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->Fy_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->Fz_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->conversion_factor_fyz = 1.0f / 1000.f;    // convert to m

    //set lattice data

    Constants->U_lb = 0.09f;                  // max 0.1 (Book suggests max 0.2)

    // set simulation parameters

    Constants->time = 8.0f;                      //[s]
    Constants->plot_every = 0.1f;               //[s]
    Constants->err_every = 0.001f;              //[s]
    Constants->iterationsMomentAvg = 10000;      //[1]

    //----------------------LOADING MESH------------------------------//

    outputerMesh::MeshMatrixIn(Data, Constants, "lesMeshSmall", 1);

    //----------------------MESHING GEOMETRY--------------------------//

    timerMeshingBoundary.start();
        Mesher.meshingBoundaryWall(0);
        Mesher.meshingBoundaryConditionInletParaboloidRectangle( cuboidInlet, inletCenter, inletDimX, inletDimY, inletDimZ, NormalInlet, meanVelocityInlet, 1 );
        //Mesher.meshingBoundaryConditionInletUniform( cuboidInlet, NormalInlet, VelocityInlet, 0);

        Mesher.meshingBoundaryConditionOutlet(cuboidOutlet, NormalOutlet, Constants->rho_fyz,
                                                1); //TODO: if density = -1 then density is from nod itself
        Mesher.compileBoundaryArrayInlets(1);
        Mesher.compileBoundaryArrayOutlets(1);
        Mesher.arrayTransfer(1);
    timerMeshingBoundary.stop();


    //----------------------MESHING OUTPUT--------------------------//


    outputerVTK::MeshVTK(Data, Constants, "meshIN");



    //----------------------SOLVING PROBLEM------------------------//


    Solver.convertToLattice(1);
    Solver.initializeSimulation(1);

    //printf("Tau - %f.", Constants -> tau );

    if(runSim) {
        Solver.runSimulation();
    }

    //----------------------TIMERS OUTPUT--------------------------//


    logger.writeHeader("Timing of sections 1) Whole loop");
    //logger.writeSystemInformation(true);
    Solver.timer_loop.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Collision");
    Solver.timer_collision.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Streaming");
    Solver.timer_streaming.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Bounce back");
    Solver.timer_bounceback.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Moments Update");
    Solver.timer_momentsUpdate.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Error Calculation");
    Solver.timer_err.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Writting Output");
    Solver.timer_output.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("TimeDumping");
    Solver.timer_dumping.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Time Averaging");
    Solver.timer_timeAvg.writeLog(logger, 0);


    return 0;
}


