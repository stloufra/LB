#include "hip/hip_runtime.h"
#pragma hd_warning_disable
#pragma nv_exec_check_disable


#include <iostream>
#include <TNL/Timer.h>
#include <TNL/Logger.h>

#include "src/geometry/geometryMesherBoundary.h"
#include "src/geometry/geometryObjectCuboid.h"
#include "src/solvers/Solver.h"
#include "src/traits/LBMTraits.h"
#include "src/postprocesors/outputerVTK.h"
#include "src/postprocesors/outputerMesh.h"

#include "src/solvers/models/D3Q27/D3Q27.h"
#include "src/solvers/models/D3Q19/D3Q19.h"
#include "src/solvers/models/D3Q15/D3Q15.h"


using namespace TNL;
using DeviceType = typename LBMTraits::DeviceType;
using VectorType = typename LBMTraits::VectorType;
using DeviceTypeHost = typename LBMTraits::DeviceTypeHost;
using LBMDataPointer = TNL::Pointers::SharedPointer<LBMData, DeviceType>;
using LBMConstantsPointer = TNL::Pointers::SharedPointer<LBMConstants, DeviceType>;

int main() {


    //------------------------INITIALIZATION--------------------------//


    // initialize data carrier objects
    LBMConstantsPointer Constants;
    LBMDataPointer Data;

    // model types selection
    using Model = D3Q15;

    using Initialisation = InitializationEquilibriumVariables<Model>;
    using Collision = CollisionSRT<Model>;
    using Streaming = StreamingAB<Model>;
    using BounceBackWall = BounceBackWallHalf<Model>;
    using Inlet = InletVelocity<Model>;
    using Outlet = OutletDensityEquilibrium<Model>;
    using Moments  = MomentDensityVelocityN15<Model>;  // SAME AS MODEL NUMBER
    using Error = ErrorQuadratic<Model>;
    using NonDim = NonDimensiolnaliseFactorsVelocity<Model>;


    //initialize timers
    Timer timerMeshingBoundary;
    Logger logger(50, std::cout);

    //initialize methodical classes
    geometryMesherBoundary Mesher(Constants,
                                  Data);

    Solver< Model,
            Initialisation,
            Collision,
            Streaming,
            BounceBackWall,
            Inlet,
            Outlet,
            Moments,
            Error,
            NonDim> Solver( Constants,
                            Data);


    //------------------------DATA IN--------------------------//

    //set simulation initialization
    VectorType Init(1.f, 0.f, 0.f);
    Constants->VelocityInit = Init;
    Constants->InitFileName = "variablesLattice199-backup-2-2-factor";


    //set meshing data
    Constants->resolution_factor = 3.f;                              // needs to be 1 or greater integer
    Constants->additional_factor = 2.f;                              // at least 1 for additional wall around
    Constants->point_outside = {0.f, 0.f, 20.f};
    Constants->file_name = "Dummy.off";

    //set geometry objects

    //resolution 3
    geometryObjectCuboid cuboidInlet1({15.f, 160.f, -80.f},
                                      {-15.f, 160.f, -80.f},
                                      {15.f, 120.f, -79.5f},
                                      3);

    geometryObjectCuboid cuboidInlet2({15.f, 200.f, 15.f},
                                      {-15.f, 200.f, 15.f},
                                      {15.f, 199.5f, -5.f},
                                      4);

    geometryObjectCuboid cuboidOutlet({15.f, 0.f, 15.f},
                                      {-15.f, 0.f, 15.f},
                                      {15.f, 0.3f, -15.f},
                                      5);


    VectorType VelocityInlet1(0.f, 0.f, 0.1f);
    VectorType VelocityInlet2(0.f, -0.2f, 0.f);
    VectorType NormalInlet1(0.f, 0.f, -1.f);
    VectorType NormalInlet2(0.f, 1.f, 0.f);
    VectorType NormalOutlet(0.f, -1.f, 0.f);


    //set physical data
    Constants->rho_fyz = 1000.f;                      //[kg/m3]
    Constants->ny_fyz = 10e-5f;                       //[m2/s]
    Constants->u_guess_fyz = 0.5f;                   //[m/s] //TODO should be automatically calculated
    Constants->Fx_fyz = 10.f;                         //[kg/m3/s2]  <- force density
    Constants->Fy_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->Fz_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->conversion_factor_fyz = 1.0f / 1000.f;    // convert to m

    //set lattice data

    Constants->U_lb = 0.09f;                  // max 0.1 (Book suggests max 0.2)

    // set simulation parameters

    Constants->time = 2.f;               //[s]
    Constants->plot_every = 0.01f;         //[s]
    Constants->err_every = 0.002f;         //[s]

    //----------------------LOADING MESH------------------------------//

    outputerMesh::MeshMatrixIn(Data, Constants, "mesh", 1);

    //----------------------MESHING GEOMETRY--------------------------//

    timerMeshingBoundary.start();
    Mesher.meshingBoundaryWall(0);
    Mesher.meshingBoundaryConditionInlet(cuboidInlet1, NormalInlet1, VelocityInlet1, 1);
    Mesher.meshingBoundaryConditionInlet(cuboidInlet2, NormalInlet2, VelocityInlet2, 1);
    Mesher.meshingBoundaryConditionOutlet(cuboidOutlet, NormalOutlet, 1000.f,
                                          1); //if density - 1 then density is from noditself
    Mesher.compileBoundaryArrayInlets(1);
    Mesher.compileBoundaryArrayOutlets(1);
    Mesher.arrayTransfer(1);
    timerMeshingBoundary.stop();


    //----------------------MESHING OUTPUT--------------------------//


    outputerVTK::MeshVTK(Data, Constants, "meshIN");

    //----------------------SOLVING PROBLEM------------------------//

    Solver.convertToLattice(1);
    Solver.initializeSimulation(1);
    Solver.runSimulation();

    //----------------------TIMERS OUTPUT--------------------------//


    logger.writeHeader("Timing of sections 1) Whole loop");
    //logger.writeSystemInformation(true);
    Solver.timer_loop.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Collision");
    Solver.timer_collision.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Streaming");
    Solver.timer_streaming.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Bounce back");
    Solver.timer_bounceback.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Moments Update");
    Solver.timer_momentsUpdate.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Error Calculation");
    Solver.timer_err.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Writting Output");
    Solver.timer_output.writeLog(logger, 0);


    return 0;
}


