#include "hip/hip_runtime.h"
#pragma hd_warning_disable
#pragma nv_exec_check_disable


#include <iostream>
#include <TNL/Timer.h>
#include <TNL/Logger.h>

#include "src/geometry/geometryMesherBoundary.h"
#include "src/geometry/geometryObjectCuboid.h"
#include "src/solvers/Solver.h"
#include "src/traits/LBMTraits.h"
#include "src/postprocesors/outputerVTK.h"
#include "src/postprocesors/outputerMesh.h"
#include "src/solvers/models/D3Q27.h"

using namespace TNL;
using DeviceType = typename LBMTraits::DeviceType;
using VectorType = typename LBMTraits::VectorType;
using DeviceTypeHost = typename LBMTraits::DeviceTypeHost;
using LBMDataPointer = TNL::Pointers::SharedPointer<LBMData, DeviceType>;
using LBMConstantsPointer = TNL::Pointers::SharedPointer<LBMConstants, DeviceType>;

int main() {


    //------------------------INITIALIZATION--------------------------//


    // initialize data carrier objects
    LBMConstantsPointer Constants;
    LBMDataPointer Data;

    //TODO: using Initialization = initializationEqulibrium;
    using ModelData = D3Q27Data;
    using Model = D3Q27<ModelData>; //TODO <Initialization >;
    //initialize timers
    Timer timerMeshingBoundary;
    Logger logger(50, std::cout);

    //initialize methodical classes


    geometryMesherBoundary Mesher(Constants,
                                  Data);

    Solver<Model, ModelData> Solver(Constants,
                  Data);
    //------------------------DATA IN--------------------------//

    //set meshing data
    Constants->resolution_factor = 3;                              // needs to be 1 or greater integer
    Constants->additional_factor = 2;                              // at least 1 for additional wall around
    Constants->point_outside = {0, 0, 20};
    Constants->file_name = "Dummy.off";

    //set geometry objects

    //resolution 3
    geometryObjectCuboid cuboidInlet1({15, 160, -80},
                                      {-15, 160, -80},
                                      {15, 120, -79.5},
                                      3);

    geometryObjectCuboid cuboidInlet2({15, 200, 15},
                                      {-15, 200, 15},
                                      {15, 199.5, -5},
                                      4);

    geometryObjectCuboid cuboidOutlet({15, 0, 15},
                                      {-15, 0, 15},
                                      {15, 0.3, -15},
                                      5);



    VectorType VelocityInlet1( 0, 0, 0.1);
    VectorType VelocityInlet2(0, -0.2, 0);
    VectorType NormalInlet1(0, 0, -1);
    VectorType NormalInlet2(0, 1, 0);
    VectorType NormalOutlet(0, -1, 0);

    VectorType VelocityInit( 0, 0, 0);

    //set physical data
    Constants->rho_fyz = 1000.f;                      //[kg/m3]
    Constants->ny_fyz = 10e-5f;                       //[m2/s]
    Constants->u_guess_fyz = 0.5f;                   //[m/s]
    Constants->Fx_fyz = 10.f;                         //[kg/m3/s2]  <- force density
    Constants->Fy_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->Fz_fyz = 0.0f;                         //[kg/m3/s2]  <- force density
    Constants->conversion_factor_fyz = 1.0 / 1000;    // convert to m

    //set lattice data

    Constants->U_lb = 0.09;                  // max 0.1 (Book suggests max 0.2)

    // set simulation parameters

    Constants -> time =0.01f;               //[s]
    Constants -> plot_every=0.001f;         //[s]

    //----------------------LOADING MESH------------------------------//

    outputerMesh::MeshMatrixIn(Data, Constants, "mesh", 1);

    //----------------------MESHING GEOMETRY--------------------------//

    timerMeshingBoundary.start();
        Mesher.meshingBoundaryWall(0);
        Mesher.meshingBoundaryConditionInlet(cuboidInlet1, NormalInlet1, VelocityInlet1, 1);
        Mesher.meshingBoundaryConditionInlet(cuboidInlet2, NormalInlet2, VelocityInlet2, 1);
        Mesher.meshingBoundaryConditionOutlet(cuboidOutlet, NormalOutlet, 0.788, 1); //if density - 1 then density is from noditself
        Mesher.compileBoundaryArrayInlets(1);
        Mesher.compileBoundaryArrayOutlets(1);
        Mesher.arrayTransfer(1);
    timerMeshingBoundary.stop();


    //----------------------MESHING OUTPUT--------------------------//


        outputerVTK::MeshVTK(Data, Constants, "meshIN");


    //----------------------SOLVING PROBLEM------------------------//

        Solver.convertToLattice(1);
        Solver.initializeSimulation(VelocityInit, 1);
        //outputerVTK::variablesLatticeVTK(Data, Constants, -1, 1);
        Solver.runSimulation();
        outputerVTK::distributionFunctionVTK(Data, Constants, 3, 1);

    //----------------------TIMERS OUTPUT--------------------------//


    logger.writeHeader("Timing of sections 1) Whole loop");
    //logger.writeSystemInformation(true);
    Solver.timer_loop.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Collision");
    Solver.timer_collision.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Streaming");
    Solver.timer_streaming.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Bounce back");
    Solver.timer_bounceback.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Moments Update");
    Solver.timer_momentsUpdate.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Error Calculation");
    Solver.timer_err.writeLog( logger, 0 );
    logger.writeSeparator();
    logger.writeHeader("Writting Output");
    Solver.timer_output.writeLog( logger, 0 );


    return 0;
}


