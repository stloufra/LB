#include "hip/hip_runtime.h"
#pragma hd_warning_disable
#pragma nv_exec_check_disable


#include <iostream>
#include <TNL/Timer.h>
#include <TNL/Logger.h>

#include "src/geometry/geometryMesherBoundary.h"
#include "src/geometry/geometryObjectCuboid.h"
#include "src/solvers/SolverTurbulentLES.h"
#include "src/traits/LBMTraits.h"
#include "src/postprocesors/outputerVTK.h"
#include "src/postprocesors/outputerMesh.h"

#include "src/solvers/models/D3Q27/D3Q27.h"
#include "src/solvers/models/D3Q19/D3Q19.h"
#include "src/solvers/models/D3Q15/D3Q15.h"


using namespace TNL;
using DeviceType = typename LBMTraits::DeviceType;
using VectorType = typename LBMTraits::VectorType;
using RealType = typename LBMTraits::RealType;
using DeviceTypeHost = typename LBMTraits::DeviceTypeHost;
using LBMDataPointer = TNL::Pointers::SharedPointer<LBMData, DeviceType>;
using LBMConstantsPointer = TNL::Pointers::SharedPointer<LBMConstants, DeviceType>;

int main() {

    //------------------------INITIALIZATION--------------------------//

    bool runSim = true;

    // initialize data carrier objects
    LBMConstantsPointer Constants;
    LBMDataPointer Data;

    // model types selection
    using Model = D3Q27;

    using Initialisation        = InitializationEquilibriumConstVector<Model>;
    using Collision             = CollisionCumD3Q27Turbulent2015<Model>;
    using Streaming             = StreamingAB<Model>;
    using BounceBackWall        = BounceBackWallHalf<Model>;
    using Symmetry              = NoSymmetry<Model>;

    using Inlet                 = InletVelocityEquilibrium<Model>;
    using Outlet                = OutletDensityInterpolated<Model>;
    using Moments               = MomentDensityVelocityN27<Model>;  // SAME AS MODEL NUMBER
    using Error                 = ErrorQuadratic<Model>;
    using Turbulence            = OmegaLES<Model>;
    using NonDim                = NonDimensiolnaliseFactorsVelocity<Model>;
    using TimeAvg               = MomentTimeAvg<Model>;


    //initialize timers
    Timer timerMeshingBoundary;
    Logger logger(50, std::cout);

    //initialize methodical classes
    geometryMesherBoundary Mesher(Constants,
                                  Data);

    SolverTurbulentLES< Model,
            Initialisation,
            Collision,
            Streaming,
            BounceBackWall,
            Symmetry,
            Inlet,
            Outlet,
            Moments,
            Turbulence,
            Error,
            NonDim,
            TimeAvg> Solver( Constants,
                            Data);



    //------------------------DATA IN--------------------------//

    //set simulation initialization
    VectorType Init(0.f, 0.f, 3.f); //change to 1 in z
    Constants->VelocityInit = Init;


   /* //set meshing data
    Constants->resolution_factor = 0.1;
    Constants->additional_factor = 6;                              // at least 1 for additional wall around
    Constants->point_outside = {2, 1000, 0};
    Constants->file_name = "Fany.off";*/


    //set geometry objects -1 streaming from it | no-bounce back - INLET
    //set geometry objects -2 streaming from and into it | bounce back - OUTLET

    //resolution 3
    geometryObjectCuboid cuboidInlet({-0.099f, 0.15f, -0.01f},
                                      {-0.099f, -0.01f, 0.11f},
                                      {-0.11, 0.15f, -0.01f},-1);


    geometryObjectCuboid cuboidOutlet({0.4575f, 0.15f, -0.01f},
                                      {0.4575f, -0.01f, 0.11f},
                                      {0.46f, 0.15f, -0.01f},-2);


    VectorType NormalInlet(-1.f, 0.f, 0.f);

    VectorType velocityInletUniform(46.78f, 0.f, 0.f);

    VectorType NormalOutlet(1.f, 0.f, 0.f);

    //inlet parabolic data

    d3 inletCenter = {-0.1f , 0.075, 0.06};
    RealType inletDimX = 0.f;
    RealType inletDimY = 0.15f;
    RealType inletDimZ = 0.1f;
    RealType meanVelocityInlet = 46.78f;       // 5

    //dumping tau outlet data
    Constants -> omegaDumpingLow = 0.f;
    Constants -> omegaDumpingHigh = 0.05f; // tau(Re=1000) = 0.55 -> 0.2


    //set physical data
    Constants->rho_fyz = 1.293f;                        //[kg/m3]     1000
    Constants->ny_fyz = 2*10e-5f;                       //[m2/s]
    Constants->u_guess_fyz = 4.f*meanVelocityInlet;     //[m/s]
    Constants->Fx_fyz = 0.0f;                           //[kg/m3/s2]  <- force density
    Constants->Fy_fyz = 0.0f;                           //[kg/m3/s2]  <- force density
    Constants->Fz_fyz = 0.0f;                           //[kg/m3/s2]  <- force density
    Constants->conversion_factor_fyz = 1.0f;            // convert to m

    //set lattice data

    Constants->U_lb = 0.09f;                  // max 0.1 (Book suggests max 0.2)

    // set simulation parameters

    Constants->time = 80.0f;                      //[s]
    Constants->plot_every = 0.5f;               //[s]
    Constants->err_every = 0.001f;              //[s]
    Constants->iterationsMomentAvg = 10000;      //[1]

    // set sampling parameters
    Constants->probe_every_it = 100;
    VectorType Probe(0.301f, 0.075f, 0.05f);
    Constants->ProbeLocation = Probe;

    //----------------------LOADING MESH------------------------------//

    outputerMesh::MeshMatrixIn(Data, Constants, "BackwardStepTurbulent", 1);

    //----------------------MESHING GEOMETRY--------------------------//

    timerMeshingBoundary.start();
        Mesher.meshingBoundaryWall(0);
        //Mesher.meshingBoundaryConditionInletParaboloidRectangle( cuboidInlet, inletCenter, inletDimX, inletDimY, inletDimZ, NormalInlet, meanVelocityInlet, 1 );


        Mesher.meshingBoundaryConditionInletUniform(cuboidInlet, NormalInlet, velocityInletUniform,0);
        Mesher.meshingBoundaryConditionOutlet(cuboidOutlet, NormalOutlet, Constants->rho_fyz,
                                                1); //TODO: if density = -1 then density is from nod itself
        Mesher.compileBoundaryArrayWall(1);
        Mesher.compileBoundaryArrayInlets(1);
        Mesher.compileBoundaryArrayOutlets(1);
        Mesher.arrayTransfer(1);
    timerMeshingBoundary.stop();


    //----------------------MESHING OUTPUT--------------------------//


    outputerVTK::MeshVTK(Data, Constants, "meshIN");



    //----------------------SOLVING PROBLEM------------------------//


    Solver.convertToLattice(1);
    Solver.initializeSimulation(1);

    if(runSim) {
        Solver.runSimulation();
    }

    //----------------------TIMERS OUTPUT--------------------------//


    logger.writeHeader("Timing of sections 1) Whole loop");
    //logger.writeSystemInformation(true);
    Solver.timer_loop.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Collision");
    Solver.timer_collision.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Streaming");
    Solver.timer_streaming.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Bounce back");
    Solver.timer_bounceback.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Moments Update");
    Solver.timer_momentsUpdate.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Error Calculation");
    Solver.timer_err.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Writting Output");
    Solver.timer_output.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("TimeDumping");
    Solver.timer_dumping.writeLog(logger, 0);
    logger.writeSeparator();
    logger.writeHeader("Time Averaging");
    Solver.timer_timeAvg.writeLog(logger, 0);


    return 0;
}
